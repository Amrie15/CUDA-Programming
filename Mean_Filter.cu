#include "hip/hip_runtime.h"
#include <stdio.h>
#include <stdlib.h>
#include <math.h>
#include <iostream>
#include <hip/hip_runtime.h>

// host input
int *a, *b;
// Host output
int *c;

__global__ void vecAdd(int *A, int *B, int*C){
    int index=threadIdx.x + blockIdx.x * blockDim.x;

    C[index]=A[index]+B[index];

}



int main(int argc, char**argv)
{
    int n = 20;
    
    

    int nBytes= n*sizeof(int);
    a=(int *)malloc(nBytes);
    b=(int *)malloc(nBytes);
    c=(int *)malloc(nBytes);

    for (int i=0; i<20;i++){
        a[i]=i+1;
        b[i]=i+1;
    }
    for (int i=0; i<20;i++){
        printf("%d \n", a[i]);
    }
    
    int *a_d, *b_d, *c_d;

    hipMalloc((void**)&a_d, nBytes);
    hipMalloc((void**)&b_d, nBytes);
    hipMalloc((void**)&c_d, nBytes);

    
    hipMemcpy(a_d,a,n*sizeof(int),hipMemcpyHostToDevice);
   hipMemcpy(b_d,b,n*sizeof(int),hipMemcpyHostToDevice);

    vecAdd<<<1, 20>>>(a_d, b_d, c_d);
    hipDeviceSynchronize();

    hipMemcpy(c, c_d, n*sizeof(int), hipMemcpyDeviceToHost);
    hipMemcpy(dest, src, bytes, hipMemcpyHostToDevice);
    

    for (int i=0;i<20;i++){
        printf("%d \n", c[i]);
    }

    
    hipFree(a_d);
    hipFree(b_d);
    hipFree(c_d);
}